#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <torch/extension.h>



void int8_matmul_cuda(torch::Tensor A, torch::Tensor B, torch::Tensor C) {

    hipblasHandle_t handle;
    hipblasCreate(&handle);
//
    int row_a = A.sizes()[0];
    int col_a = A.sizes()[1];
    int row_b = B.sizes()[0];
    int col_b = B.sizes()[1];

    int alpha = 1;
    int beta = 0;

    hipblasGemmEx(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, row_a, row_b, col_a,
                  &alpha, A.data_ptr<int8_t>(), HIP_R_8I, col_a, B.data_ptr<int8_t>(), HIP_R_8I, col_b, &beta,
                  C.data_ptr<int32_t>(), HIP_R_32I, row_a, HIPBLAS_COMPUTE_32I, HIPBLAS_GEMM_DEFAULT);
    hipblasDestroy(handle);

}
